#include "hip/hip_runtime.h"
#ifdef _WIN32
#include <windows.h>
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "VoxelDataStructures.h"
#include "VoxelUtils.h"
#include "common.h"

//This is a simple vector math library. Use this with CUDA instead of glm
#include "cuda_helper/cuda_SimpleMatrixUtil.h"

#define FREE_BLOCK -1
#define LOCKED_BLOCK -2
#define NO_OFFSET 0

__constant__ HashTableParams d_hashtableParams;
__constant__ float3x3 kinectProjectionMatrix;
__constant__ PtrContainer d_ptrHldr;
PtrContainer h_ptrHldr;


__inline__ __device__
bool FIRST_THREAD()	{
	if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)	{
		return true;
	}
	return false;
}

__inline__ __device__
void printDeviceMatrix(const float3x3& mat)	{

	printf("Printing device matrix...\n");
	for(int i=0;i<3;++i)	{
		for(int j=0;j<3;++j)	{
			printf("%f\t", mat.entries2[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

/*----------(Raw)STORAGE---------------*/
//VoxelEntry *d_hashTable;
//VoxelEntry *d_compactifiedHashTable;
//int* d_compactifiedHashCounter;
//unsigned int *d_compactifiedHashCounter;
//int *d_heap;	//Arena that manages free memory
//int *d_heapCounter;	//single element; points to next free block (atomic counter)
//__device__ int d_heapCounter;
//Voxel *d_SDFBlocks;
//int *d_hashTableBucketMutex;	//mutex for locking particular bin while inserting/deleting
/*------------------------------------*/

/*---------(Thrust)STORAGE--------------*/

//hashtable
//thrust::device_vector<VoxelEntry> d_hashTable_vec;
//thrust::device_vector<VoxelEntry> d_compactifiedHashTable_vec;
	//thrust::device_vector<int> d_hashTableBucketMutex_vec;
	//__device__ int d_compactifiedHashCounter;

	//VoxelEntry* d_hashTable;
	//VoxelEntry* d_compactifiedHashTable;
	//int* d_hashTableBucketMutex;

	//heap management
	//thrust::device_vector<int> d_heap_vec;	//heap that manages free memory
	//__device__ int d_heapCounter;

	//int* d_heap;

	//actual voxelblocks
	//thrust::device_vector<Voxel> d_SDFBlocks_vec;	//main heap holding tsdf blocks

	//Voxel* d_SDFBlocks;
	/*------------------------------*/

	//! Make rigid transform available on the device
	void updateConstantHashTableParams(const HashTableParams &params)	{
		size_t size;
		checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(d_hashtableParams)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_hashtableParams), &params, size, 0, hipMemcpyHostToDevice));
	}

	void updateDevicePointers() {
		size_t size;
		checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(d_ptrHldr)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_ptrHldr), &h_ptrHldr, size, 0, hipMemcpyHostToDevice));
		std::cout << "h_ptrHldr supposedly copied to device\n";
	}

	//__host__
	//void allocate(const HashTableParams& params)	{
	//	const int initVal = 0;
	//	d_hashTable_vec.resize(params.numBuckets * params.bucketSize);
	//	d_compactifiedHashTable_vec.resize(params.numBuckets * params.bucketSize);
	//	d_hashTableBucketMutex_vec.resize(params.numBuckets * params.bucketSize);
	//	d_heap_vec.resize(params.numBuckets * params.bucketSize);
	//	d_SDFBlocks_vec.resize(params.numBuckets * params.bucketSize * 512);
	//	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ptrHldr.d_heapCounter), &initVal, sizeof(int),
	//			0, hipMemcpyHostToDevice));
	//	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ptrHldr.d_compactifiedHashCounter), &initVal,
	//			sizeof(int), 0, hipMemcpyHostToDevice));
	//	//init raw pointers
	//	ptrHldr.d_heap = thrust::raw_pointer_cast(&d_heap_vec[0]);
	//	ptrHldr.d_hashTable = thrust::raw_pointer_cast(&d_hashTable_vec[0]);
	//	ptrHldr.d_compactifiedHashTable = thrust::raw_pointer_cast(&d_compactifiedHashTable_vec[0]);
	//	ptrHldr.d_hashTableBucketMutex = thrust::raw_pointer_cast(&d_hashTableBucketMutex_vec[0]);
	//	ptrHldr.d_SDFBlocks = thrust::raw_pointer_cast(&d_SDFBlocks_vec[0]);
	//}

	//TODO : Confusion here. FInish this later
	//no GL functions should be called after pointers are mapped to cuda
	//__host__
	extern "C" void mapGLobjectsToCUDApointers(hipGraphicsResource* numBlocks_res, hipGraphicsResource* compactHashtable_res,
		hipGraphicsResource* sdfVolume_res) {

		size_t returnedBufferSize;
		checkCudaErrors(hipGraphicsMapResources(1, &numBlocks_res, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&h_ptrHldr.d_compactifiedHashCounter, &returnedBufferSize, numBlocks_res));
		int occupiedBlocks = -1;
		//TODO : following line is just a check. remove it later
		checkCudaErrors(hipMemcpy(&occupiedBlocks, &h_ptrHldr.d_compactifiedHashCounter[0], sizeof(int), hipMemcpyDeviceToHost));
		std::cout << "(after GL mapping)numVisibleBlocks : " << occupiedBlocks << "\n";

		checkCudaErrors(hipGraphicsMapResources(1, &compactHashtable_res, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&h_ptrHldr.d_compactifiedHashTable, &returnedBufferSize, compactHashtable_res));
		std::cout << "(after GL mapping) size of compactifiedHashtable(in bytes) : " << returnedBufferSize<< "\n";

		checkCudaErrors(hipGraphicsMapResources(1, &sdfVolume_res, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&h_ptrHldr.d_SDFBlocks, &returnedBufferSize, sdfVolume_res));
		std::cout << "(after GL mapping) size of SDFVoxelBlocks(in bytes) : " << returnedBufferSize << "\n";
		updateDevicePointers();

	}

	//__host__
	extern "C" void resetHashTableMutexes(const HashTableParams& params) {
		checkCudaErrors(hipMemset(h_ptrHldr.d_hashTableBucketMutex, 0, sizeof(int)*params.numBuckets));
		updateDevicePointers();
	}

	__global__
	void resetHashTableKernel(VoxelEntry* table) {
		const int idx = blockIdx.x*blockDim.x + threadIdx.x;
		if (idx >= d_hashtableParams.bucketSize*d_hashtableParams.numBuckets) return;
		table[idx].offset = NO_OFFSET;
		table[idx].ptr = FREE_BLOCK;
		table[idx].pos = make_int3(INF, INF, INF);
	}

	//Call this only once!!!
	__global__
	void resetHeapKernel(unsigned int* d_heap) {
		const int idx = blockIdx.x*blockDim.x + threadIdx.x;
		if (idx >= d_hashtableParams.numVoxelBlocks) return;
		d_heap[idx] = idx;
	}

	//TODO do this using thrust instead
	__host__
	void deviceAllocate(const HashTableParams &params)	{
		//PtrContainer h_ptrHldr;
		checkCudaErrors(hipMalloc((void**)&h_ptrHldr.d_heap, sizeof(unsigned int) * params.numVoxelBlocks));
		checkCudaErrors(hipMalloc((void**)&h_ptrHldr.d_hashTable, sizeof(VoxelEntry) * params.numBuckets * params.bucketSize));
		//checkCudaErrors(hipMalloc((void**)&h_ptrHldr.d_compactifiedHashTable, sizeof(VoxelEntry) * params.numBuckets * params.bucketSize));
		checkCudaErrors(hipMalloc((void**)&h_ptrHldr.d_hashTableBucketMutex, sizeof(int) * params.numBuckets));
		//checkCudaErrors(hipMalloc((void**)&h_ptrHldr.d_SDFBlocks, sizeof(Voxel) * params.numVoxelBlocks * params.voxelBlockSize * params.voxelBlockSize * params.voxelBlockSize));
		checkCudaErrors(hipMalloc((void**)&h_ptrHldr.d_heapCounter, sizeof(int)));
		//checkCudaErrors(hipMalloc((void**)&h_ptrHldr.d_compactifiedHashCounter, sizeof(int)));	//TODO : remove this

		updateDevicePointers();

		//init with correct values
		const int totalThreads = params.numBuckets*params.bucketSize;
		int blocks = (totalThreads / 1024) + 1;
		int threads = 1024;
		//init buffers with default values. TODO: Launch following two kernels asynchronously
		resetHashTableKernel<<<blocks, threads >>> (h_ptrHldr.d_hashTable);
		checkCudaErrors(hipDeviceSynchronize());
		//TODO : reset compactifiedHashTable after registering GL buffers
		//resetHashTableKernel<<<blocks, threads >>> (h_ptrHldr.d_compactifiedHashTable);
		//checkCudaErrors(hipDeviceSynchronize());

		int heapBlocks = (params.numVoxelBlocks / threads) + 1;
		resetHeapKernel<<<heapBlocks, threads>>>(h_ptrHldr.d_heap);
		checkCudaErrors(hipDeviceSynchronize());

		//set rest of data 0
		//checkCudaErrors(hipMemset(h_ptrHldr.d_heap, 0, sizeof(int)*params.numVoxelBlocks));	//don't need this anymore
		checkCudaErrors(hipMemset(h_ptrHldr.d_hashTableBucketMutex, 0, sizeof(int)*params.numBuckets));
		//TODO : reset SDFBlocks after registering GL buffers
		//checkCudaErrors(hipMemset(h_ptrHldr.d_SDFBlocks, 0, sizeof(Voxel) * params.numVoxelBlocks *
			//params.voxelBlockSize * params.voxelBlockSize * params.voxelBlockSize));
		checkCudaErrors(hipMemset(h_ptrHldr.d_heapCounter, 0, sizeof(int)));
		//checkCudaErrors(hipMemset(h_ptrHldr.d_compactifiedHashCounter, 0, sizeof(int)));	//TODO : remove this

		//set d_heapCounter = numVoxelBlocks -1;
		int heapCounterInitVal = params.numVoxelBlocks - 1;
		checkCudaErrors(hipMemcpy(&h_ptrHldr.d_heapCounter[0], &heapCounterInitVal, sizeof(int), hipMemcpyHostToDevice));
		//now copy this struct back to device
		updateDevicePointers();
	}

	__host__
	void deviceFree()	{
		checkCudaErrors(hipFree(d_ptrHldr.d_hashTable));
		checkCudaErrors(hipFree(d_ptrHldr.d_heap));
		checkCudaErrors(hipFree(d_ptrHldr.d_heapCounter));
		//checkCudaErrors(hipFree(d_ptrHldr.d_compactifiedHashTable));
		//checkCudaErrors(hipFree(d_ptrHldr.d_compactifiedHashCounter));	//TODO : remove this
		//checkCudaErrors(hipFree(d_ptrHldr.d_SDFBlocks));
		checkCudaErrors(hipFree(d_ptrHldr.d_hashTableBucketMutex));
	}

	__host__
	void calculateKinectProjectionMatrix()	{

		float3x3 m(intrinsics);
		//Now upload to device
		std::cout<<"Uploading projection matrix to device..\n";
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kinectProjectionMatrix), &m, sizeof(m)));
	}

	//TODO : Remove this function later
	__inline__ __device__
	bool vertexInFrustum(float4 point)	{
		point = d_hashtableParams.global_transform * point;
		float3 pos = make_float3(point.x, point.y, point.z);
		pos = kinectProjectionMatrix * pos;
		pos = pos/pos.z;	//normalize, and get screen coordinates
		int x = __float2int_rz(pos.x);
		int y = __float2int_rz(pos.y);
		if(x < 640 && x >=0 && y < 480 && y >= 0)	{
			return true;
		}
		return false;
	}


	//Now actual GPU code
	__device__
	unsigned int calculateHash(const int3& pos)	{
			const int p0 = 73856093;
			const int p1 = 19349669;
			const int p2 = 83492791;

			int res = ((pos.x * p0) ^ (pos.y * p1) ^ (pos.z * p2)) % d_hashtableParams.numBuckets;
			if (res < 0) res += d_hashtableParams.numBuckets;
			return (uint)res;
	}

	__device__
	float getTruncation(float z)	{
		return d_hashtableParams.truncation + (d_hashtableParams.truncScale * z);
	}

	__device__
	int3 voxel2Block(int3 voxel) 	{
		const int size = d_hashtableParams.voxelBlockSize;
		//float3 vx = make_float3(voxel);
		//int x = __float2int_rz(vx.x / size);
		//int y = __float2int_rz(vx.y / size);
		//int z = __float2int_rz(vx.z / size);
		//return make_int3(x, y, z);
		if(voxel.x < 0) voxel.x -= size-1;	//i.e voxelBlockSize -1
		if(voxel.y < 0) voxel.y -= size-1;
		if(voxel.z < 0) voxel.z -= size-1;
		return make_int3(voxel.x/size, voxel.y/size, voxel.z/size);
	}

	__device__
	int3 world2Voxel(const float3& point)	{
		const float size = d_hashtableParams.voxelSize;
		float3 p = point/size;
		int3 centerOffset = make_int3(copysignf(1, p.x), copysignf(1, p.y), copysignf(1, p.z));
		int3 voxelPos =  make_int3(p + make_float3(centerOffset.x*0.5, centerOffset.y*0.5, centerOffset.z*0.5));//return center
		return voxelPos;
	}

	__device__
	int3 block2Voxel(const int3& block)	{
		int3 voxelPos = make_int3(block.x, block.y, block.z) * d_hashtableParams.voxelBlockSize;
		return voxelPos;
	}

	__device__
	float3 voxel2World(const int3& voxel)	{
		float3 worldPos = make_float3(voxel) * d_hashtableParams.voxelSize;
		return worldPos;
	}

	__device__
	float3 block2World(const int3& block)	{
		return voxel2World(block2Voxel(block));
	}

	__device__
	int3 world2Block(const float3& point)	{
		return voxel2Block(world2Voxel(point));
	}

	__device__
	unsigned int linearizeVoxelPos(const int3& pos)	{
		const int size = d_hashtableParams.voxelBlockSize;
		return  pos.z * size * size +
				pos.y * size +
				pos.x;
	}

	__device__
	int3 delinearizeVoxelPos(const unsigned int index)	{
		const int size = d_hashtableParams.voxelBlockSize;
		unsigned int x = index % size;
		unsigned int y = (index % (size * size)) / size;
		unsigned int z = index / (size * size);
		return make_int3(x,y,z);
	}

	__inline__ __device__
	int allocSingleBlockInHeap()	{	//int ptr
		//decrement total available blocks by 1
		int addr = atomicSub(&d_ptrHldr.d_heapCounter[0], 1);	//TODO: make this uint
		//if (addr < 0) return -1;	//negative index shouldn't, but still happens :(
		return d_ptrHldr.d_heap[addr];
	}

	__device__
	void removeSingleBlockInHeap(int ptr)	{
		//int delIdx = ptr / 512;
		int addr = atomicAdd(&d_ptrHldr.d_heapCounter[0], 1);	//TODO: make this uint
		d_ptrHldr.d_heap[addr + 1] = ptr;
	}

	//Frustum culling
	__inline__ __device__
	bool blockInFrustum(int3 blockId) {
		//return true;
		float3 worldPos = block2World(blockId);
		float4 pos = make_float4(worldPos.x, worldPos.y, worldPos.z, 1);
		pos = d_hashtableParams.global_transform * pos;	//TODO : shouldn't this be inv_global_transform?
		float3 projected = make_float3(pos.x, pos.y, pos.z);
		projected = kinectProjectionMatrix * projected;
		projected = projected / projected.z;

		int x = __float2int_rz(projected.x);
		int y = __float2int_rz(projected.y);
		if (x < 640 && x >= 0 && y < 480 && y >= 0) {
			return true;
		}
		return false;
	}

	//Hacky but cool code below
	__device__
	VoxelEntry getVoxelEntry4Block(const int3& pos)	{
		const unsigned int hash = calculateHash(pos);
		const unsigned int bucketSize = d_hashtableParams.bucketSize;
		const unsigned int numBuckets = d_hashtableParams.numBuckets;
		const unsigned int startIndex = hash * bucketSize;

		VoxelEntry temp;
		temp.pos = pos;
		temp.offset = 0;
		temp.ptr = FREE_BLOCK;

		int i=0;
		//[1] Iterate all bucketSize entries
		for(i=0; i < bucketSize ; ++i)	{
			VoxelEntry& curr = d_ptrHldr.d_hashTable[startIndex + i];
			if((curr.pos.x == pos.x) && (curr.pos.y == pos.y) &&(curr.pos.z == pos.z)
					&& (curr.ptr != FREE_BLOCK)) {
				return curr;
			}
		}

#ifdef LINKED_LIST_ENABLED

		//[2] block not found. handle collisions by traversing tail linked list
		const int lastEntryInBucket = (hash+1)*bucketSize -1;
		i = lastEntryInBucket;
		//memorize idx at list end and memorize offset from last
		//element of bucket to list end
		int iter = 0;
		const int maxIter = d_hashtableParams.attachedLinkedListSize;
		while(iter < maxIter)	{

			VoxelEntry curr = d_ptrHldr.d_hashTable[i];
			if((curr.pos.x == pos.x) && (curr.pos.y == pos.y) &&(curr.pos.z == pos.z)
					&& (curr.ptr != FREE_BLOCK)) {
				return curr;
			}

			if(curr.offset == 0)	{ //we've found end of list
				break;
			}
			i = lastEntryInBucket + curr.offset;

			i %= (numBuckets * bucketSize);

			iter++;
		}

#endif // LINKED_LIST_ENABLED

		return temp;
	}


	//TODO incomplete function
	__inline__ __device__
	bool insertVoxelEntry(const int3& data)	{

		unsigned int hash = calculateHash(data);
		const unsigned int bucketSize = d_hashtableParams.bucketSize;
		const unsigned int numBuckets = d_hashtableParams.numBuckets;
		const unsigned int startIndex = hash * bucketSize;

		VoxelEntry temp;
		temp.offset=0;
		temp.ptr = FREE_BLOCK;
		temp.pos = data;

		//[1] iterate current bucket, try inserting at first empty block we see.
		int i=0;
		if(FIRST_THREAD())	{
			printf("Insertion : before bucket iteration\n");
		}
		for(i=0; i<bucketSize; ++i)	{
			int idx = startIndex+i;
			idx = idx % (numBuckets * bucketSize);
			VoxelEntry &curr = d_ptrHldr.d_hashTable[idx];
			if(curr.pos.x == data.x && curr.pos.y == data.y && curr.pos.z == data.z
					&& curr.ptr != FREE_BLOCK)	return false;
			if(curr.ptr == FREE_BLOCK)	{
				//TODO shouldn't the following be [hash] instead of [idx] ?
				int prevVal = atomicExch(&d_ptrHldr.d_hashTableBucketMutex[hash], LOCKED_BLOCK);
				if(prevVal != LOCKED_BLOCK)	{	//means we can lock current bucket
				//{
					curr.pos = data;
					curr.offset = NO_OFFSET;
					int ptrIdx = allocSingleBlockInHeap() * 512;
					if (ptrIdx < 0)	return false;	//all VoxelBlocks occupied
					curr.ptr = ptrIdx;
					//printf("Inserted block : (%d, %d, %d) at idx %d\n", data.x, data.y, data.z, ptrIdx/512);
					return true;
				}
			}
		}

#ifdef LINKED_LIST_ENABLED

		//[2] bucket is full. Append to list.
		const int lastEntryInBucket = (hash+1)*bucketSize - 1;

		i = lastEntryInBucket;
		int offset=0;
		//memorize idx at list end and memorize offset from last
		//element of bucket to list end
		int iter = 0;
		const int maxIter = d_hashtableParams.attachedLinkedListSize;
		while(iter < maxIter)	{
			i = i%(numBuckets*bucketSize);
			VoxelEntry& curr = d_ptrHldr.d_hashTable[i];
			if(curr.ptr != FREE_BLOCK)	{
				if(curr.pos.x == data.x && curr.pos.y == data.y &&
						curr.pos.z == data.z && curr.ptr != FREE_BLOCK)	{
					return false;	//alloc unsuccessful because block already there
				}
				if(curr.offset == 0)	{//end of list, lookahead till we find empty slot
					int j=1;
					//[1] lock the parent block
					int prevVal = atomicExch(&d_ptrHldr.d_hashTableBucketMutex[hash],
							LOCKED_BLOCK);
					if(prevVal != LOCKED_BLOCK)	{//if we got the lock
						//[2] then lookahead for empty block in new bucket
						while(j<10)	{
							if(d_ptrHldr.d_hashTable[i+j].ptr == FREE_BLOCK)	break;
							j++;
						}
						if(j==10)	{
							//we couldn't find empty space despite looking ahead 10 spaces
							return false;
						}
						//[3] now lock this new bucket and insert the block
						prevVal = atomicExch(&d_ptrHldr.d_hashTableBucketMutex[(i+j)/numBuckets],
								LOCKED_BLOCK);
						if(prevVal != LOCKED_BLOCK)	{
							VoxelEntry& next = d_ptrHldr.d_hashTable[i+j];
							//TODO maybe we can do away with this check
							if(next.ptr == FREE_BLOCK)	{
								int ptrIdx = allocSingleBlockInHeap() * 512;
								if (ptrIdx < 0)	return false;
								next.ptr = ptrIdx;
								next.pos = data;
								curr.offset = j;
								break;
							}
							i++;
						}
					}
				}
				//TODO: implement this correctly
				if(curr.offset != 0)	{	//traversing nodes in linked list
					int j = i;
					while(j <= (i+curr.offset))	{
						if(d_ptrHldr.d_hashTable[j].ptr == FREE_BLOCK)	{
							//[a] free space found. first lock bucket with curr
							int prevVal = atomicExch(&d_ptrHldr.d_hashTableBucketMutex[hash/numBuckets], LOCKED_BLOCK);
							if(prevVal != LOCKED_BLOCK)	{
								//[b] then lock bucket with new space
								prevVal = atomicExch(&d_ptrHldr.d_hashTableBucketMutex[j/numBuckets], LOCKED_BLOCK);
								if(prevVal != LOCKED_BLOCK)	{
									VoxelEntry& ins = d_ptrHldr.d_hashTable[j];
									ins.offset = i + curr.offset - j;
									int ptrIdx = allocSingleBlockInHeap() * 512;
									if (ptrIdx < 0)	return false;
									ins.ptr = ptrIdx;
									ins.pos = data;
									curr.offset = j - i;
									return true;
								}
							}
						}
						j++;
					}
					i += curr.offset;
				}
			}
			iter++;
		}
#endif // LINKED_LIST_ENABLED

	}


	__device__
	bool deleteVoxelEntry(int3 data)	{
		//TODO : iterate over entire bucket
		unsigned int hash = calculateHash(data);
		const unsigned int bucketSize = d_hashtableParams.bucketSize;
		const unsigned int numBuckets = d_hashtableParams.numBuckets;
		const unsigned int startIndex = hash * bucketSize;

		VoxelEntry temp;
		temp.offset=0;
		temp.ptr = FREE_BLOCK;
		temp.pos = data;

		//[1] iterate current bucket, try inserting at first empty block we see.
		int i=0;
		for(i=0; i<bucketSize; ++i)	{
			const int idx = startIndex+i;
			VoxelEntry &curr = d_ptrHldr.d_hashTable[idx];
			if(curr.pos.x == data.x && curr.pos.y == data.y && curr.pos.z == data.z
					&& curr.ptr != FREE_BLOCK)	{return false;}
			if(curr.ptr == FREE_BLOCK)	{
				//TODO shouldn't the following be [hash] instead of [idx] ?
				//try locking current bucket
				int prevVal = atomicExch(&d_ptrHldr.d_hashTableBucketMutex[hash], LOCKED_BLOCK);
				if(prevVal != LOCKED_BLOCK)	{	//means we can lock current bucket
					curr.pos = make_int3(0);
					curr.offset = NO_OFFSET;
					removeSingleBlockInHeap(curr.ptr/512);
					curr.ptr = FREE_BLOCK;
					return true;
				}
			}
		}

#ifdef LINKED_LIST_ENABLED

		//deletion in linked list
		int lastEntry = beforeThis(data);
		if(lastEntry == -1)	{return false;}	//error
		VoxelEntry& prev = d_ptrHldr.d_hashTable[lastEntry];
		VoxelEntry& curr = d_ptrHldr.d_hashTable[lastEntry + prev.offset];
		//lock the bucket with curr
		int prevVal = atomicExch(&d_ptrHldr.d_hashTableBucketMutex[hash], LOCKED_BLOCK);
		if(prevVal!=LOCKED_BLOCK)	{	//lock acquired
			prevVal = atomicExch(&d_ptrHldr.d_hashTableBucketMutex[lastEntry / numBuckets],
					LOCKED_BLOCK);
			if(prevVal != LOCKED_BLOCK)	{
				//TODO FINISH THIS!!!
				prev.offset += curr.offset;
				curr.pos = make_int3(0);
				curr.offset = NO_OFFSET;
				removeSingleBlockInHeap(curr.ptr/512);
				curr.ptr = FREE_BLOCK;
			}
		}

		return false;//delete didn't happen :(

#endif // LINKED_LIST_ENABLED

	}

	__global__
	void allocBlocksKernel(const float4* verts, const float4* normals)	{	//Do we need normal data here?

		const float voxelSize = d_hashtableParams.voxelSize;
		int xidx = blockDim.x*blockIdx.x + threadIdx.x;
		int yidx = blockDim.y*blockIdx.y + threadIdx.y;

		if (xidx >= numCols || yidx >= numRows) {
			return;
		}

		//find globalIdx row-major
		const int idx = (yidx*numCols) + xidx;

		float4 tempPos = verts[idx];
		if(tempPos.z == 0.0f) return;
		tempPos = d_hashtableParams.global_transform * tempPos;	//transform to global frame
		//float3 projTemp = make_float3(tempPos.x, tempPos.y, tempPos.z);
		//projTemp = kinectProjectionMatrix * projTemp;
		//projTemp = projTemp/projTemp.z;
		//TODO : Erase this later
		//if(idx==153600)	{
		//	printf("Middle vertex (%f, %f, %f, %f)\n",verts[idx].x, verts[idx].y, verts[idx].z, verts[idx].w);
		//}
		float3 p = make_float3(tempPos);
		float3 pn = make_float3(normals[idx]);
		float3 rayStart = p;// -(d_hashtableParams.truncation * pn);
		//float3 rayEnd = p + (d_hashtableParams.truncation * pn);

		//Now find their voxel blocks
		//check if block is in view, then insert into table
		int3 startBlock = world2Block(rayStart);
		if(blockInFrustum(startBlock))	{	//blockInFrustum(temp)
			insertVoxelEntry(startBlock);
		}	//, instead simply

		//By now all necessary blocks should have been allocated
	}

	//! Allocate all hash blocks which are corresponding to depth map entries
	extern "C" void allocBlocks(const float4* verts, const float4* normals)	{

		const dim3 blocks(640/16, 480/16, 1);
		//const dim3 blocks(1, 1, 1);
		const dim3 threads(16, 16, 1);
		std::cout<<"Running AllocBlocksKernel\n";
		allocBlocksKernel <<<blocks, threads>>>(verts, normals);
		checkCudaErrors(hipDeviceSynchronize());
	}

	//! Generate a linear hash-array with only occupied entries
	__global__
	void flattenKernel()	{
		const int idx = (blockDim.x * blockIdx.x) + threadIdx.x;

		if(idx >= (d_hashtableParams.bucketSize * d_hashtableParams.numBuckets)) return;

		__shared__ int localCounter;
		if(threadIdx.x == 0) localCounter = 0;
		__syncthreads();

		//local address within block
		int localAddr = -1;
		const VoxelEntry& entry = d_ptrHldr.d_hashTable[idx];
		if(entry.ptr != FREE_BLOCK && blockInFrustum(entry.pos))	{
			localAddr = atomicAdd(&localCounter, 1);
		}
		__syncthreads();

		//update global count of occupied blocks
		__shared__ int globalAddr;
		if(threadIdx.x==0 && localCounter > 0)	{
			globalAddr = atomicAdd(&d_ptrHldr.d_compactifiedHashCounter[0], localCounter);
		}
		__syncthreads();

		//assign local address and copy
		if(localAddr != -1)	{
			const unsigned int addr = globalAddr + localAddr;
			d_ptrHldr.d_compactifiedHashTable[addr] = entry;
		}
	}

	extern "C" int flattenIntoBuffer(const HashTableParams& params)	{
		//first set numOccupiedBlocks = 0
		//first clear previously flattened hashtable buffer
		const int totalThreads = params.numBuckets*params.bucketSize;
		int blocks = (totalThreads / 1024) + 1;
		int threads = 1024;
		//TODO : Do we really need to reset compactifiedHashTable? wouldn't it get overwritten by flattenKernel anyways?
		resetHashTableKernel <<<blocks, threads >>> (h_ptrHldr.d_compactifiedHashTable);
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipMemset(h_ptrHldr.d_compactifiedHashCounter, 0, sizeof(int)));
		checkCudaErrors(hipDeviceSynchronize());

		flattenKernel<<<blocks, threads>>>();
		checkCudaErrors(hipDeviceSynchronize());
		int occupiedBlocks = 0;
		checkCudaErrors(hipMemcpy(&occupiedBlocks, &h_ptrHldr.d_compactifiedHashCounter[0], sizeof(int), hipMemcpyDeviceToHost));

		return occupiedBlocks;
	}

	__inline__ __device__
	int2 project(float3 voxelWorldPos) {
		//int3 pos = make_int3(point.x, point.y, point.z);
		//float3 worldPos = voxel2World(voxel);
		voxelWorldPos = kinectProjectionMatrix * voxelWorldPos;
		voxelWorldPos = voxelWorldPos / voxelWorldPos.z;
		return make_int2(voxelWorldPos.x, voxelWorldPos.y);
	}

	__inline __device__
	Voxel combineVoxel(const Voxel& oldVox, const Voxel& currVox) {
		//TODO: add color later
		Voxel newVox;
		//supposedly 'correct' code - doesn't produce right viz tho
		//newVox.weight = min(d_hashtableParams.integrationWeightMax, (float)oldVox.weight + (float)currVox.weight);
		//newVox.sdf = ((oldVox.sdf * (float)oldVox.weight) + (currVox.sdf * (float)currVox.weight)) / (newVox.weight);

		//old, wrong code -> but right viz
		newVox.sdf = ((oldVox.sdf * (float)oldVox.weight) + (currVox.sdf * (float)currVox.weight)) / ((float)oldVox.weight + (float)currVox.weight);
		newVox.weight = min(d_hashtableParams.integrationWeightMax, (float)oldVox.weight + (float)currVox.weight);
		return newVox;
	}

	//Implementation of Curless & Levoy paper(1996)
	__global__
	void integrateDepthMapKernel(const float4* verts) {

		/*
		//Testing : draw SDF sphere to ensure this integration kernel is working ok
		const VoxelEntry& entry = d_ptrHldr.d_compactifiedHashTable[blockIdx.x];
		int3 base_voxel = block2Voxel(entry.pos);
		int3 i = make_int3(threadIdx.x, threadIdx.y, threadIdx.z);
		int3 curr_voxel = base_voxel + i;
		int3 i_temp = make_int3(-4, -4, -4); i_temp = i_temp + i;
		float sdf = (i_temp.x * i_temp.x) + (i_temp.y * i_temp.y) + (i_temp.z * i_temp.z) - (3*3); //sphere
		//int sign = signbit(sdf) ? 1 : -1;
		//sdf = sqrt(sdf)*sign;
		sdf = sqrt(sdf);
		const float temp_truncation_val = 4.0;

		if (abs(sdf) < temp_truncation_val)
		{
			//float weightUpdate = fmaxf(temp_truncation_val - (abs(sdf)/temp_truncation_val), 1.0f); //ie more weight when near 0
			float weightUpdate = 0.1f;
			Voxel curr;
			curr.sdf = sdf;
			curr.weight = weightUpdate;
			//curr.color = make_uchar3(0, 255, 0);	//TODO : later

			int linIdx = linearizeVoxelPos(i);
			const int oldVoxIdx = entry.ptr + linIdx;
			const Voxel oldVox = d_ptrHldr.d_SDFBlocks[oldVoxIdx];
			Voxel fusedVoxel = combineVoxel(oldVox, curr);
			//printf("(%f, %f)", fusedVoxel.sdf, fusedVoxel.weight);	//Working fine till here
			d_ptrHldr.d_SDFBlocks[oldVoxIdx] = fusedVoxel;	//replace old voxel with new fused one

		}
		*/
		//-------------------------------------------------------
		const VoxelEntry& entry = d_ptrHldr.d_compactifiedHashTable[blockIdx.x];
		int3 base_voxel = block2Voxel(entry.pos);

		int3 i = make_int3(threadIdx.x, threadIdx.y, threadIdx.z);
		int3 curr_voxel = base_voxel + i;// delinearizeVoxelPos(i);
		float4 curr_voxel_float = make_float4(curr_voxel.x, curr_voxel.y, curr_voxel.z, 1.0);
		curr_voxel_float = d_hashtableParams.inv_global_transform * curr_voxel_float;
		curr_voxel = make_int3(curr_voxel_float.x, curr_voxel_float.y, curr_voxel_float.z);

		float3 voxel_worldPos = voxel2World(curr_voxel);
		int2 screenPos = project(voxel_worldPos);

		if ((screenPos.x < 0) || (screenPos.x >= 640) || (screenPos.y < 0) || (screenPos.y >= 480)) return;
		const int idx = (screenPos.y * 640) + screenPos.x;

		float inDepth = verts[idx].z; //depth from depth-map
		if (inDepth <= 0)	return;

		//TODO : define these explicitly, somewhere safe outside of here!
		float depthRangeMin = 0.5;	//metres
		float depthRangeMax = 5.0;
		float depthZeroOne = (inDepth - depthRangeMin) / (depthRangeMax - depthRangeMin);	//normalize current depth

		//assert(voxel_worldPos.z > 0)
		float sdf = inDepth - voxel_worldPos.z;
		//printf("%f", sdf);	//Working fine till here
		float truncation = d_hashtableParams.truncation;	// +(d_hashtableParams.truncScale*depth);
		//i.e calculate truncation of the SDF for given depth value

		if (abs(sdf) < abs(truncation)) {
			if (sdf >= 0.0f) {
				sdf = fminf(truncation, sdf);
			}
			else {
				sdf = fmaxf(-truncation, sdf);
			}

			//Sets updation weight based on sensor noise. Farther depths have less weight. Copied from prof. Niessner's implementation
			//float weightUpdate = fminf(d_hashtableParams.integrationWeightSample * (1.0 - depthZeroOne), 1.0f);
			float weightUpdate = fminf(d_hashtableParams.integrationWeightSample * (depthZeroOne) * d_hashtableParams.voxelSize, 1.0f);
			//float weightUpdate = fmaxf(d_hashtableParams.integrationWeightSample * (depthZeroOne), 1.0f);
			//unsigned int weightUpdate = 10;	//let's keep this constant for now
			//float  weightUpdate = 0.2f;

			Voxel curr;
			curr.sdf = sdf;
			curr.weight = weightUpdate;
			//curr.color = make_uchar3(0, 255, 0);	//TODO : later

			const int oldVoxIdx = entry.ptr + linearizeVoxelPos(i);
			const Voxel oldVox = d_ptrHldr.d_SDFBlocks[oldVoxIdx];
			Voxel fusedVoxel = combineVoxel(oldVox, curr);
			//printf("(%f, %f)", fusedVoxel.sdf, fusedVoxel.weight);	//Working fine till here
			d_ptrHldr.d_SDFBlocks[oldVoxIdx] = fusedVoxel;	//replace old voxel with new fused one
		}
	}

	extern "C" void integrateDepthMap(const HashTableParams& params, const float4* verts) {
		dim3 threads = dim3(params.voxelBlockSize, params.voxelBlockSize, params.voxelBlockSize);
		int blocks = params.currentOccupiedBlocks;

	if (params.currentOccupiedBlocks > 0) {
		integrateDepthMapKernel<<<blocks, threads>>>(verts);
		checkCudaErrors(hipDeviceSynchronize());
	}
}
