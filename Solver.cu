#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
//#include "Solver.h"
#include "cuda_helper/hip/hip_vector_types.h"
#include <thrust/fill.h>
#include <thrust/device_ptr.h>

#define numCols 640

const int pyr_size = 3;
std::array<dim3, pyr_size> solver_blocks  = {dim3(20, 15), dim3(20, 15), dim3(20, 15)}; //, dim3(16, 12)};
std::array<dim3, pyr_size> solver_threads = {dim3(32, 32), dim3(16, 16), dim3(8,   8)}; //, dim3(5, 5)}	;

//using FloatVec = thrust::device_vector<float>;
//using Float4Vec = thrust::device_vector<float4>;

//using CorrPairVec = thrust::device_vector<CorrPair>;

__device__ inline
float CalculateResidual(const float3& n, const float3& d, const float3& s)
{
  float3 p = (d - s);
  return (dot(p,n));
}

__device__ inline
void CalculateJacobians(float* JacMat, const float3& d, const float3& n, int index)
{
  float3 T = (cross(d, n));
  // Calculate Jacobian for this correspondence pair. Probably most important piece
  // of code in entire project
  JacMat[index*6]     = n.x;
  JacMat[index*6 + 1] = n.y;
  JacMat[index*6 + 2] = n.z;
  JacMat[index*6 + 3] = T.x;
  JacMat[index*6 + 4] = T.y;
  JacMat[index*6 + 5] = T.z;
  //JacMat.row(index) << n.x, n.y, n.z, T.x, T.y, T.z ;
}

__global__
void CalculateJacAndResKernel(const float4* d_src, const float4* d_dest, const float4* d_destNormals,float* d_JacMat,
															const int width, const int height)
{
	int xidx = blockDim.x*blockIdx.x + threadIdx.x;
	int yidx = blockDim.y*blockIdx.y + threadIdx.y;
	//find globalIdx row-major
	const int idx = (yidx*width) + xidx;
  float3 src = make_float3(d_src[idx]);
  float3 dest = make_float3(d_dest[idx]);
  float3 destNormal = make_float3(d_destNormals[idx]);
  CalculateJacobians(d_JacMat, dest, destNormal, idx);
  //residual[idx] = pair.distance;
}

//__device__ inline
//void CalculateJTJ

extern "C" void CalculateJacobiansAndResiduals(const float4* d_src, const float4* d_targ, const float4* d_targNormals,
    float* d_Jac, float* d_residuals, const int pyrLevel, const int width, const int height)
{

  //First calculate Jacobian and Residual matrices
  //float4* d_targ = thrust::raw_pointer_cast(&targ[0]);
  //float4* d_targNormals = thrust::raw_pointer_cast(&targNormals[0]);
  //float* d_jacobianMatrix = thrust::raw_pointer_cast(&Jac[0]);
  //float* d_resVector = thrust::raw_pointer_cast(&residual[0]);
  //float* d_jtj = thrust::raw_pointer_cast(&JTJ[0]);
  //float* d_jtr = thrust::raw_pointer_cast(&JTr[0]);
	int numCorrPairs = width*height; //number of correspondence pairs
  thrust::device_ptr<float> d_Jac_ptr = thrust::device_pointer_cast(d_Jac);
  thrust::fill(d_Jac_ptr, d_Jac_ptr+(numCorrPairs*6), 0);  //TODO - is this redundant?
  CalculateJacAndResKernel<<<solver_blocks[pyrLevel], solver_threads[pyrLevel]>>>(d_src, d_targ, d_targNormals, d_Jac,
			width, height);

  //Then calculate Matrix-vector JTr and Matrix-matrix JTJ products
}

